
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//__global__声明该函数为需要在GPU上计算的核函数
__global__ void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main()
{
    int N = 1 << 20;
    float *x, *y;

    //在GPU上开辟内存
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // 初始化
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // 在GPU上计算,调用256个thread
    add<<<1, 256>>>(N, x, y);

    // 在访问Host之前,先等待GPU的运算结束
    hipDeviceSynchronize();

    //检查误差:数组y所有的值都应该为3
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error:" << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);

    return 0;
}